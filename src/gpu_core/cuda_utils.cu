#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include <iostream>

namespace CudaUtils {

void printDeviceInfo() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found!" << std::endl;
        return;
    }
    
    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, dev));
        
        std::cout << "Device " << dev << ": " << prop.name << std::endl;
        std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Total global memory: " << prop.totalGlobalMem / (1024*1024*1024) << " GB" << std::endl;
        std::cout << "  Shared memory per block: " << prop.sharedMemPerBlock / 1024 << " KB" << std::endl;
        std::cout << "  Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "  Warp size: " << prop.warpSize << std::endl;
        std::cout << "  Memory clock rate: " << prop.memoryClockRate / 1000 << " MHz" << std::endl;
        std::cout << "  Memory bus width: " << prop.memoryBusWidth << " bits" << std::endl;
    }
}

void allocateDeviceMemory(float** d_ptr, size_t size) {
    CUDA_CHECK(hipMalloc(d_ptr, size * sizeof(float)));
}

void allocateDeviceMemory(int** d_ptr, size_t size) {
    CUDA_CHECK(hipMalloc(d_ptr, size * sizeof(int)));
}

void allocateDeviceMemory(hiprandState** d_ptr, size_t size) {
    CUDA_CHECK(hipMalloc(d_ptr, size * sizeof(hiprandState)));
}

void copyToDevice(float* d_ptr, const float* h_ptr, size_t size) {
    CUDA_CHECK(hipMemcpy(d_ptr, h_ptr, size * sizeof(float), hipMemcpyHostToDevice));
}

void copyToHost(float* h_ptr, const float* d_ptr, size_t size) {
    CUDA_CHECK(hipMemcpy(h_ptr, d_ptr, size * sizeof(float), hipMemcpyDeviceToHost));
}

void copyToDevice(int* d_ptr, const int* h_ptr, size_t size) {
    CUDA_CHECK(hipMemcpy(d_ptr, h_ptr, size * sizeof(int), hipMemcpyHostToDevice));
}

void copyToHost(int* h_ptr, const int* d_ptr, size_t size) {
    CUDA_CHECK(hipMemcpy(h_ptr, d_ptr, size * sizeof(int), hipMemcpyDeviceToHost));
}

dim3 calculateGridDim(int width, int height, int blockSizeX, int blockSizeY) {
    int gridX = (width + blockSizeX - 1) / blockSizeX;
    int gridY = (height + blockSizeY - 1) / blockSizeY;
    return dim3(gridX, gridY);
}

dim3 calculateBlockDim(int blockSizeX, int blockSizeY) {
    return dim3(blockSizeX, blockSizeY);
}

dim3 calculate1DGridDim(int numElements, int blockSize) {
    int numBlocks = (numElements + blockSize - 1) / blockSize;
    return dim3(numBlocks);
}

dim3 calculate1DBlockDim(int blockSize) {
    return dim3(blockSize);
}

void freeDeviceMemory(void* d_ptr) {
    if (d_ptr != nullptr) {
        CUDA_CHECK(hipFree(d_ptr));
    }
}

} // namespace CudaUtils
